#include "hip/hip_runtime.h"
#include "global.h"
// This file is used to parse the input data and get it to the form that is required.
void parseCSVData(float values[N_SAMPLES][N_FEATURES]){
    /*
     * This subroutine parses data from an input file defined by INPUT.
     * It parses a csv file and extracts x, y, z, vx, vy, vz, m in the same order & discards an id field defined in the input file.
     */
    FILE* stream;
    int idx = 0;
    stream = fopen(INPUT, "r");
    for(idx = 0; fscanf(stream,"%f,%f,%f,%f,%f,%f,%f,%*f\n",
                        &values[idx][INDEX_X],
                        &values[idx][INDEX_Y],
                        &values[idx][INDEX_Z],
                        &values[idx][INDEX_VX],
                        &values[idx][INDEX_VY],
                        &values[idx][INDEX_VZ],
                        &values[idx][INDEX_M]) != EOF && idx < N_SAMPLES; idx++);
    return;
}
void generate3DData(float values[N_SAMPLES][N_FEATURES], int x[2], int y[2], int z[2], int vx[2], int vy[2], int vz[2], int m[2]){
    /*
     * This subroutine generates 3-dimensional data given the min and max values for spatial coordinates x, y, z, velocities vx, vy, vz and mass m.
     */
    return;
}
/*
 *  Usage example:
 *
 *  int main(){
 *      float arr[N_SAMPLES][N_FEATURES];
 *      int i;
 *      parseCSVData(arr);
 *      // CODE USING `arr`
 *      return 0;
 *  }
 *
 */
