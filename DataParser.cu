#include "hip/hip_runtime.h"
#include "global.h"
// This file is used to parse the input data and get it to the form that is required.
void parseCSVData(float values[N][FEATURES]){
    /*
     * This subroutine parses data from an input file defined by INPUT.
     * It parses a csv file and extracts x, y, z, vx, vy, vz, m in the same order & discards an id field defined in the input file.
     */
    FILE* stream;
    int idx = 0;
    stream = fopen(INPUT, "r");
    for(idx = 0; fscanf(stream,"%f,%f,%f,%f,%f,%f,%f,%*f\n",
                        &values[idx][INDEX_X],
                        &values[idx][INDEX_Y],
                        &values[idx][INDEX_Z],
                        &values[idx][INDEX_VX],
                        &values[idx][INDEX_VY],
                        &values[idx][INDEX_VZ],
                        &values[idx][INDEX_M]) != EOF && idx < N; idx++);
    return;
}
void generate3DData(float values[N][FEATURES], int x[2], int y[2], int z[2], int vx[2], int vy[2], int vz[2], int m[2]){
    /*
     * This subroutine generates 3-dimensional data given the min and max values for spatial coordinates x, y, z, velocities vx, vy, vz and mass m.
     */
    return;
}
int main(){
    float arr[N][FEATURES];
    int i;
    printf("Hello");
    parseCSVData(arr);
    printf("Hello");
    for(i = 0; i < N; i++){
        printf("%d %f\n",i,arr[i][0]);
    }
    return 0;
}
